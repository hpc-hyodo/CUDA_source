//CPU���g�����x�N�g�����Z

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 10240
#define M 100000

void addVector(const float *a, const float *b, float *c)
{
	int i;
	for (i = 0; i < N; i++){
		c[i] = a[i] + b[i];
	}
}

int main(void)
{
	float *a;
	float *b;
	float *c;
	int i;
	unsigned int begin, end;

	srand(time(NULL));

	a = (float *)malloc(N*sizeof(float));
	b = (float *)malloc(N*sizeof(float));
	c = (float *)malloc(N*sizeof(float));

	for (i = 0; i < N; i++){
		a[i] = rand() / (float)RAND_MAX;
		b[i] = rand() / (float)RAND_MAX;
	}

	begin = clock();
	for (i = 0; i < M; i++){
		addVector(a, b, c);
	}
	end = clock();
	for (i = 0; i < N; i++){
		if (c[i] != a[i] + b[i]){
			printf("error\n");
			break;
		}
	}
	
	printf("�v�Z���Ԃ�%d(ms)�ł�\n", end - begin);

	free(a);
	free(b);
	free(c);

	return 0;
}
