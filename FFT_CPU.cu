#include <stdio.h>
#include "FFT.h"
#define N 16 /* N��2�ׂ̂��� */

// 7787285617673431 * 100712375473872�̌v�Z 
void main(void)
{
	int i, j, rgsi, cy = 0, radix = 10000, ip[N / 2 + 2], ans[N + 2];
	double scale, a[N + 2], b[N + 2], c[N + 2], w[N / 2 + 2], rgs, gr;

	for (i = 0; i<N; i++) a[i] = 0.0;
	a[N - 4] = 7787.0;
	a[N - 3] = 2856.0;
	a[N - 2] = 1767.0;
	a[N - 1] = 3431.0;

	for (i = 0; i<N; i++) b[i] = 0.0;
	b[N - 4] = 100.0;
	b[N - 3] = 7123.0;
	b[N - 2] = 7547.0;
	b[N - 1] = 3872.0;

	ip[0] = 0;
	rdft(N, 1, a, ip, w);
	rdft(N, 1, b, ip, w);

	c[0] = a[0] * b[0];
	c[1] = a[1] * b[1];
	for (i = 2; i<N; i += 2){
		c[i] = a[i] * b[i] - a[i + 1] * b[i + 1];
		c[i + 1] = a[i + 1] * b[i] + a[i] * b[i + 1];
	}

	rdft(N, -1, c, ip, w);

	scale = 2.0 / N;
	gr = 1.0 / radix;
	ans[0] = 0;
	for (i = N - 1; i>0; i--){
		rgs = c[i - 1] * scale + 0.5 + cy;
		cy = rgs*gr;
		rgsi = rgs;
		ans[i] = rgsi - cy*radix;
	}

	for (i = 0; i<N; i++) printf("%04d", ans[i]);
	printf("\n");
}
