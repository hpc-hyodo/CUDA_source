//CPU�s���Z�@1024*1024

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <time.h>

#define N 1024

void multMatrix(const float *a, const float *b, float *c)
{
	int i, j, k;

	for (i = 0; i < N; i++){
		for (j = 0; j < N; j++){
			float v;
			v = 0.;
			for (k = 0; k < N; k++){
				v += a[i*N + k] * b[k*N + j];
			}
			c[i*N + j] = v;
		}
	}
}

int main(void)
{
	float *a;
	float *b;
	float *c;
	int i, j, k;
	unsigned int begin, end;

	srand(time(NULL));

	a = (float *)malloc(N*N*sizeof(float));
	b = (float *)malloc(N*N*sizeof(float));
	c = (float *)malloc(N*N*sizeof(float));

	for (i = 0; i < N*N; i++){
		a[i] = rand() / (float)RAND_MAX;
		b[i] = rand() / (float)RAND_MAX;
	}
	begin = clock();
	multMatrix(a, b, c);

	for (i = 0; i < N; i++){
		for (j = 0; j < N; j++){
			float v;
			v = 0.;
			for (k = 0; k < N; k++){
				v += a[i*N + k] * b[k*N + j];
			}
			if (c[i*N + j] != v){
				printf("error\n");
				goto loop_end;
			}
		}
	}
loop_end:
	end = clock();
	for (i = 0; i < N*N; i++){
		printf("c[%d] = %f \n", i, c[i]);
	}
	printf("�v�����Ԃ�%d�ł�", end - begin);

	free(a);
	free(b);
	free(c);

	return 0;
}