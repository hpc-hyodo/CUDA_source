#include "hip/hip_runtime.h"
//GPU����v�Z�@�s��̏�Z 1024*1024
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 1024

#define BLOCK_SIZE 32


struct Matrix
{
	int width;
	int height;
	float *elements;
};

__global__ void multMatrix(const float *a, const float *b, float *c)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	int k;
	float v;

	v = 0.;
	for (k = 0; k < N; k++){
		v += a[i*N + k] * b[k*N + j];
		__syncthreads();
	}

	c[i*N + j] = v;
}


int main(void)
{
	float *a;
	float *b;
	float *c;
	float *a_d;
	float *b_d;
	float *c_d;
	dim3 grid;
	dim3 block;
	int i, j, k;
	unsigned int begin, end, begin2, end2;

	srand(time(NULL));

	hipHostMalloc(&a, N*N*sizeof(float));
	hipHostMalloc(&b, N*N*sizeof(float));
	hipHostMalloc(&c, N*N*sizeof(float));

	hipMalloc(&a_d, N*N*sizeof(float));
	hipMalloc(&b_d, N*N*sizeof(float));
	hipMalloc(&c_d, N*N*sizeof(float));

	for (i = 0; i < N*N; i++){
		a[i] = (double)rand()/(RAND_MAX+1);
		b[i] = (double)rand() /(RAND_MAX + 1);
	}

	hipMemcpy(a_d, a, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, N*N*sizeof(float), hipMemcpyHostToDevice);

	grid = dim3(N / BLOCK_SIZE, N / BLOCK_SIZE);
	block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	
	begin = clock();
	multMatrix << <grid, block >> >(a_d, b_d, c_d);
	end = clock();

	hipMemcpy(c, c_d, N*N*sizeof(float), hipMemcpyDeviceToHost);
	begin2 = clock();
	for (i = 0; i < N; i++){
		for (j = 0; j < N; j++){
			float v;
			v = 0.;
			for (k = 0; k < N; k++){
				v += a[i*N + k] * b[k*N + j];
			}
			if (fabs((c[i*N + j] - v) / v) > 1e-5){
				printf("error\n");
				goto loop_end;
			}
		}
	}
loop_end:
	end2 = clock();
	
	//for (i = 0; i < N*N; i++){
	//	printf("c[%d] = %.f \n", i, c[i]);
	//}
	printf("BLOCK_SIZE:%d\n", BLOCK_SIZE);
	printf("Number of block in grid�F%d\n", (N / BLOCK_SIZE)* (N / BLOCK_SIZE));
	printf("Number of thread in block�F%d\n", BLOCK_SIZE * BLOCK_SIZE);
	printf("Total Number of thread: %d\n", (BLOCK_SIZE*BLOCK_SIZE)*((N / BLOCK_SIZE)* (N / BLOCK_SIZE)));
	printf("CPU�v�Z���ԁF%d(ms)\n", end2 - begin2);
	printf("GPU�v�Z���ԁF%d(ms)", end - begin);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	return 0;
}