#include "hip/hip_runtime.h"
//GPU����v�Z�@�s��̏�Z 1024*1024
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 1024

#define BLOCK_SIZE 32


struct Matrix
{
	int width;
	int height;
	float *elements;
};

__global__ void multMatrix(const float *a, const float *b, float *c)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	int k;
	float v;

	v = 0.;
	for (k = 0; k < N; k++){
		v += a[i*N + k] * b[k*N + j];
	}

	c[i*N + j] = v;
}


int main(void)
{
	float *a;
	float *b;
	float *c;
	float *a_d;
	float *b_d;
	float *c_d;
	dim3 grid;
	dim3 block;
	int i, j, k;
	unsigned int begin, end;

	srand(time(NULL));

	hipHostMalloc(&a, N*N*sizeof(float));
	hipHostMalloc(&b, N*N*sizeof(float));
	hipHostMalloc(&c, N*N*sizeof(float));

	hipMalloc(&a_d, N*N*sizeof(float));
	hipMalloc(&b_d, N*N*sizeof(float));
	hipMalloc(&c_d, N*N*sizeof(float));

	for (i = 0; i < N*N; i++){
		a[i] = (double)rand()/(RAND_MAX+1);
		b[i] = (double)rand() /(RAND_MAX + 1);
	}

	hipMemcpy(a_d, a, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, N*N*sizeof(float), hipMemcpyHostToDevice);

	grid = dim3(N / BLOCK_SIZE, N / BLOCK_SIZE);
	block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	
	begin = clock();
	multMatrix << <grid, block >> >(a_d, b_d, c_d);
	

	hipMemcpy(c, c_d, N*N*sizeof(float), hipMemcpyDeviceToHost);

	for (i = 0; i < N; i++){
		for (j = 0; j < N; j++){
			float v;
			v = 0.;
			for (k = 0; k < N; k++){
				v += a[i*N + k] * b[k*N + j];
			}
			if (fabs((c[i*N + j] - v) / v) > 1e-5){
				printf("error\n");
				goto loop_end;
			}
		}
	}
loop_end:
	end = clock();
	for (i = 0; i < N*N; i++){
		printf("c[%d] = %.f \n", i, c[i]);
	}
	printf("�v�����Ԃ�%d�ł�", end - begin);

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	return 0;
}